#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>


using std::cout;
using std::cerr;
using std::endl;

#define N 10000000


__global__ void vecAdd(float* A, float *B, float *C, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) C[idx] = A[idx] + B[idx];
}



int main ()
{
    // pre- allocate & initialize variable on host
    float* h_A = new float[N];
    float* h_B = new float[N];
    float* h_C = new float[N];

    for (int i = 0; i < N; i++)
    {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    // 0. allocate device memory (using cudaMalloc)
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * sizeof(float));
    hipMalloc((void**)&d_B, N * sizeof(float));
    hipMalloc((void**)&d_C, N * sizeof(float));

    // 1. Copy data from host to device
    /*
    Copies data between host and device:
    cudaError_t cudaMemcpy(void *dst, const void *src, size_t count, cudaMemcpyKind kind)
    */
    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

    // 2. Run Kernel
    /* 
    2.1 define kernel execution config
        This config determines how many threads and blocks are launched to 
        execute a kernel on the GPU. This is crucial for optimizing performance.
    */
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    
    /*
    kernel<<<numBlocks, numThreads>>>(args);
    where:
        numBlocks → Number of blocks per grid
        numThreads → Number of threads per block
    */
    auto start = std::chrono::high_resolution_clock::now();
    vecAdd<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
    // Ensure all device threads finish execution
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> gpu_time = end - start;

    // 3. Copy result back to host
    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // 4. Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


    // Run CPU version for comparison
    start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < N; i++) {
        h_C[i] = h_A[i] + h_B[i];
    }
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpu_time = end - start;

    // Print performance results
    std::cout << "GPU Time: " << gpu_time.count() * 1000 << " ms\n";
    std::cout << "CPU Time: " << cpu_time.count() * 1000 << " ms\n";

    // Free memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C; 

    return 0;
}